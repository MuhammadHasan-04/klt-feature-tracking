#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "klt.h"
#include "base.h"
#include "error.h"

#define CUDA_CHECK(call)                                  \
    {                                                     \
        hipError_t err = (call);                         \
        if (err != hipSuccess)                           \
        {                                                 \
            fprintf(stderr, "CUDA Error: %s (line %d)\n", \
                    hipGetErrorString(err), __LINE__);   \
            hipDeviceReset();                            \
            exit(EXIT_FAILURE);                           \
        }                                                 \
    }

    

__global__ void featureStrengthComputation(
    const float *gradx, const float *grady, float *strength,
    int width, int height, int win_half_w, int win_half_h,
    int borderx, int bordery)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < borderx)
        return;

    if (x >= width - borderx)
        return;

    if (y < bordery)
        return;

    if (y >= height - bordery)
        return;

    float gxx = 0.0f, gxy = 0.0f, gyy = 0.0f;

    for (int j = -win_half_h; j <= win_half_h; j++)
    {
        for (int i = -win_half_w; i <= win_half_w; i++)
        {
            int idx = (y + j) * width + (x + i);
            float gxVal = gradx[idx];
            float gyVal = grady[idx];

            gxx += gxVal * gxVal;
            gxy += gxVal * gyVal;
            gyy += gyVal * gyVal;
        }
    }

    float trace = gxx + gyy;
    float diff = gxx - gyy;
    float temp = sqrtf(diff * diff + 4.0f * gxy * gxy);
    float lambda_min = 0.5f * (trace - temp);

    strength[y * width + x] = lambda_min;
}

// Wrapper function to handle GPU-based feature strength computation
extern "C" void gpuComputeFeatureStrength(
    _KLT_FloatImage gradx,
    _KLT_FloatImage grady,
    _KLT_FloatImage strength,
    int win_width,
    int win_height,
    int borderx,
    int bordery)
{
    int width = gradx->ncols;
    int height = gradx->nrows;
    size_t bufferSize = static_cast<size_t>(width * height * sizeof(float));

    float *dev_gradx = nullptr;
    float *dev_grady = nullptr;
    float *dev_strength = nullptr;

    printf("Launching GPU kernel for corner strength estimation...\n");

    CUDA_CHECK(hipMalloc(&dev_gradx, bufferSize));
    CUDA_CHECK(hipMalloc(&dev_grady, bufferSize));
    CUDA_CHECK(hipMalloc(&dev_strength, bufferSize));

    CUDA_CHECK(hipMemcpy(dev_gradx, gradx->data, bufferSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dev_grady, grady->data, bufferSize, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(
        (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipEvent_t evtStart, evtEnd;
    CUDA_CHECK(hipEventCreate(&evtStart));
    CUDA_CHECK(hipEventCreate(&evtEnd));
    CUDA_CHECK(hipEventRecord(evtStart));

    featureStrengthComputation<<<numBlocks, threadsPerBlock>>>(
        dev_gradx, dev_grady, dev_strength,
        width, height,
        win_width / 2, win_height / 2,
        borderx, bordery);

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(evtEnd));
    CUDA_CHECK(hipEventSynchronize(evtEnd));

    float gpuTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&gpuTime, evtStart, evtEnd));
    printf("GPU execution finished in %.3f ms\n", gpuTime);

    CUDA_CHECK(hipMemcpy(strength->data, dev_strength, bufferSize, hipMemcpyDeviceToHost));

    hipFree(dev_gradx);
    hipFree(dev_grady);
    hipFree(dev_strength);

    CUDA_CHECK(hipEventDestroy(evtStart));
    CUDA_CHECK(hipEventDestroy(evtEnd));
    CUDA_CHECK(hipDeviceSynchronize());
}
